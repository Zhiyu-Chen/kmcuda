#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cfloat>
#include <cinttypes>
#include <cinttypes>
#include <algorithm>
#include <memory>

#include "private.h"

#define BS_KMPP 512
#define BS_LL_ASS 256
#define BS_LL_CNT 256
#define BS_YY_INI 256
#define BS_YY_GFL 512
#define BS_YY_LFL 512
#define BLOCK_SIZE 1024  // for all the rest of the kernels

#define YINYANG_GROUP_TOLERANCE 0.02
#define YINYANG_DRAFT_REASSIGNMENTS 0.11
#define YINYANG_REFRESH_EPSILON 1e-4

#define FOR_OTHER_DEVS(...) do { \
  for (size_t odevi = 0; odevi < devs.size(); odevi++) { \
    if (odevi == devi) { \
      continue; \
    } \
    __VA_ARGS__; \
  } } while(false)

#define CUP2P(what, offset, size) do { \
  CUCH(hipMemcpyPeerAsync( \
      (*what)[odevi].get() + offset, devs[odevi], (*what)[devi].get() + offset, \
      devs[devi], size * sizeof(std::remove_reference<decltype(*what)>::type \
      ::value_type::element_type)), \
       kmcudaMemoryCopyError); \
} while(false)

__device__ uint32_t changed_number;
__device__ uint32_t passed_number;
__constant__ uint32_t samples_size;
__constant__ uint16_t features_size;
__constant__ uint32_t clusters_size;
__constant__ uint32_t yy_groups_size;
__constant__ int shmem_size;

__global__ void kmeans_plus_plus(
    const uint32_t border, const uint32_t cc, const float *__restrict__ samples,
    const float *__restrict__ centroids, float *__restrict__ dists,
    float *__restrict__ dist_sums) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= border) {
    return;
  }
  samples += static_cast<uint64_t>(sample) * features_size;
  extern __shared__ float local_dists[];
  float dist = 0;
  if (samples[0] == samples[0]) {
    uint32_t coffset = (cc - 1) * features_size;
    #pragma unroll 4
    for (uint16_t f = 0; f < features_size; f++) {
      float d = samples[f] - centroids[coffset + f];
      dist += d * d;
    }
    dist = sqrt(dist);
  }
  float prev_dist = dists[sample];
  if (dist < prev_dist || cc == 1) {
    dists[sample] = dist;
  } else {
    dist = prev_dist;
  }
  local_dists[threadIdx.x] = dist;
  uint32_t end = blockDim.x;
  if ((blockIdx.x + 1) * blockDim.x > samples_size) {
    end = samples_size - blockIdx.x * blockDim.x;
  }
  __syncthreads();
  if (threadIdx.x % 16 == 0) {
    float psum = 0;
    for (uint32_t i = threadIdx.x; i < end && i < threadIdx.x + 16; i++) {
      psum += local_dists[i];
    }
    local_dists[threadIdx.x] = psum;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    float block_sum = 0;
    for (uint32_t i = 0; i < end; i += 16) {
      block_sum += local_dists[i];
    }
    dist_sums[blockIdx.x] = block_sum;
  }
}

__global__ void kmeans_assign_lloyd(
    const uint32_t border, const float *__restrict__ samples,
    const float *__restrict__ centroids, uint32_t *__restrict__ assignments_prev,
    uint32_t * __restrict__ assignments) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= border) {
    return;
  }
  samples += static_cast<uint64_t>(sample) * features_size;
  float min_dist = FLT_MAX;
  uint32_t nearest = UINT32_MAX;
  extern __shared__ float shared_centroids[];
  const uint32_t cstep = shmem_size / (features_size + 1);
  float *csqrs = shared_centroids + cstep * features_size;
  const uint32_t size_each = cstep / blockDim.x + 1;
  bool insane = samples[0] != samples[0];
  float ssqr = 0;
  if (!insane) {
    #pragma unroll 4
    for (int f = 0; f < features_size; f++) {
      float v = samples[f];
      ssqr += v * v;
    }
  }

  for (uint32_t gc = 0; gc < clusters_size; gc += cstep) {
    uint32_t coffset = gc * features_size;
    __syncthreads();
    if (threadIdx.x * size_each < cstep) {
      for (uint32_t i = 0; i < size_each; i++) {
        uint32_t ci = threadIdx.x * size_each + i;
        uint32_t local_offset = ci * features_size;
        uint32_t global_offset = coffset + local_offset;
        if (global_offset < clusters_size * features_size) {
          float csqr = 0;
          #pragma unroll 4
          for (int f = 0; f < features_size; f++) {
            float v = centroids[global_offset + f];
            shared_centroids[local_offset + f] = v;
            csqr += v * v;
          }
          csqrs[ci] = csqr;
        }
      }
    }
    __syncthreads();
    if (insane) {
      continue;
    }
    for (uint32_t c = gc; c < gc + cstep && c < clusters_size; c++) {
      float dist = 0;
      coffset = (c - gc) * features_size;
      #pragma unroll 4
      for (int f = 0; f < features_size; f++) {
        dist += samples[f] * shared_centroids[coffset + f];
      }
      dist = ssqr + csqrs[c - gc] - 2 * dist;
      if (dist < min_dist) {
        min_dist = dist;
        nearest = c;
      }
    }
  }
  if (nearest == UINT32_MAX) {
    if (!insane) {
      printf("CUDA kernel kmeans_assign: nearest neighbor search failed for "
             "sample %" PRIu32 "\n", sample);
      return;
    } else {
      nearest = clusters_size;
    }
  }
  uint32_t ass = assignments[sample];
  assignments_prev[sample] = ass;
  if (ass != nearest) {
    assignments[sample] = nearest;
    atomicAdd(&changed_number, 1);
  }
}

__global__ void kmeans_adjust(
    const uint32_t border, const float *__restrict__ samples,
    const uint32_t *__restrict__ assignments_prev,
    const uint32_t *__restrict__ assignments, float *__restrict__ centroids,
    uint32_t *__restrict__ ccounts) {
  uint32_t c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c >= border) {
    return;
  }
  uint32_t my_count = ccounts[c];
  centroids += c * features_size;
  for (int f = 0; f < features_size; f++) {
    centroids[f] *= my_count;
  }
  extern __shared__ uint32_t ass[];
  int step = shmem_size / 2;
  for (uint32_t sbase = 0; sbase < samples_size; sbase += step) {
    __syncthreads();
    if (threadIdx.x == 0) {
      int pos = sbase;
      for (int i = 0; i < step && sbase + i < samples_size; i++) {
        ass[2 * i] = assignments[pos + i];
        ass[2 * i + 1] = assignments_prev[pos + i];
      }
    }
    __syncthreads();
    for (int i = 0; i < step && sbase + i < samples_size; i++) {
      uint32_t this_ass = ass[2 * i];
      uint32_t  prev_ass = ass[2 * i + 1];
      float sign = 0;
      if (prev_ass == c && this_ass != c) {
        sign = -1;
        my_count--;
      } else if (prev_ass != c && this_ass == c) {
        sign = 1;
        my_count++;
      }
      if (sign != 0) {
        uint64_t soffset = sbase + i;
        soffset *= features_size;
        #pragma unroll 4
        for (int f = 0; f < features_size; f++) {
          centroids[f] += samples[soffset + f] * sign;
        }
      }
    }
  }
  // my_count can be 0 => we get NaN and never use this cluster again
  // this is a feature, not a bug
  #pragma unroll 4
  for (int f = 0; f < features_size; f++) {
    centroids[f] /= my_count;
  }
  ccounts[c] = my_count;
}

__global__ void kmeans_yy_init(
    const uint32_t border, const float *__restrict__ samples,
    const float *__restrict__ centroids, const uint32_t *__restrict__ assignments,
    const uint32_t *__restrict__ groups, float *__restrict__ bounds) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= border) {
    return;
  }
  bounds += static_cast<uint64_t>(sample) * (yy_groups_size + 1);
  for (uint32_t i = 0; i < yy_groups_size + 1; i++) {
    bounds[i] = FLT_MAX;
  }
  bounds++;
  samples += static_cast<uint64_t>(sample) * features_size;
  uint32_t nearest = assignments[sample];
  extern __shared__ float shared_centroids[];
  const uint32_t cstep = shmem_size / features_size;
  const uint32_t size_each = cstep / blockDim.x + 1;

  for (uint32_t gc = 0; gc < clusters_size; gc += cstep) {
    uint32_t coffset = gc * features_size;
    __syncthreads();
    if (threadIdx.x * size_each < cstep) {
      for (uint32_t i = 0; i < size_each; i++) {
        uint32_t local_offset = (threadIdx.x * size_each + i) * features_size;
        uint32_t global_offset = coffset + local_offset;
        if (global_offset < clusters_size * features_size) {
          #pragma unroll 4
          for (int f = 0; f < features_size; f++) {
            shared_centroids[local_offset + f] = centroids[global_offset + f];
          }
        }
      }
    }
    __syncthreads();

    for (uint32_t c = gc; c < gc + cstep && c < clusters_size; c++) {
      float dist = 0;
      coffset = (c - gc) * features_size;
      uint32_t group = groups[c];
      if (group >= yy_groups_size) {
        // this may happen if the centroid is insane (NaN)
        continue;
      }
      #pragma unroll 4
      for (int f = 0; f < features_size; f++) {
        float d = samples[f] - shared_centroids[coffset + f];
        dist += d * d;
      }
      dist = sqrt(dist);
      if (c != nearest) {
        if (dist < bounds[group]) {
          bounds[group] = dist;
        }
      } else {
        bounds[-1] = dist;
      }
    }
  }
}

__global__ void kmeans_yy_calc_drifts(
    const uint32_t border, const float *__restrict__ centroids,
    float *__restrict__ drifts) {
  uint32_t c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c >= border) {
    return;
  }
  uint32_t coffset = c * features_size;
  float sum = 0;
  for (uint32_t f = coffset; f < coffset + features_size; f++) {
    float d = centroids[f] - drifts[f];
    sum += d * d;
  }
  drifts[clusters_size * features_size + c] = sqrt(sum);
}

__global__ void kmeans_yy_find_group_max_drifts(
    const uint32_t border, const uint32_t *__restrict__ groups,
    float *__restrict__ drifts) {
  uint32_t group = blockIdx.x * blockDim.x + threadIdx.x;
  if (group >= border) {
    return;
  }
  const uint32_t doffset = clusters_size * features_size;
  const uint32_t size_each = shmem_size / (2 * blockDim.x);
  const uint32_t step = size_each * blockDim.x;
  extern __shared__ uint32_t shmem[];
  float *cd = (float *)shmem;
  uint32_t *cg = shmem + shmem_size / 2;
  float my_max = FLT_MIN;
  for (uint32_t offset = 0; offset < clusters_size; offset += step) {
    __syncthreads();
    for (uint32_t i = 0; i < size_each; i++) {
      uint32_t local_offset = threadIdx.x * size_each + i;
      uint32_t global_offset = offset + local_offset;
      if (global_offset < clusters_size) {
        cd[local_offset] = drifts[doffset + global_offset];
        cg[local_offset] = groups[global_offset];
      }
    }
    __syncthreads();
    for (uint32_t i = 0; i < step; i++) {
      if (cg[i] == group) {
        float d = cd[i];
        if (my_max < d) {
          my_max = d;
        }
      }
    }
  }
  drifts[group] = my_max;
}

__global__ void kmeans_yy_global_filter(
    const uint32_t border, const float *__restrict__ samples,
    const float *__restrict__ centroids, const uint32_t *__restrict__ groups,
    const float *__restrict__ drifts, const uint32_t *__restrict__ assignments,
    uint32_t *__restrict__ assignments_prev, float *__restrict__ bounds,
    uint32_t *__restrict__ passed) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= border) {
    return;
  }
  bounds += static_cast<uint64_t>(sample) * (yy_groups_size + 1);
  uint32_t cluster = assignments[sample];
  assignments_prev[sample] = cluster;
  float upper_bound = bounds[0];
  uint32_t doffset = clusters_size * features_size;
  float cluster_drift = drifts[doffset + cluster];
  upper_bound += cluster_drift;
  bounds++;
  float min_lower_bound = FLT_MAX;
  for (uint32_t g = 0; g < yy_groups_size; g++) {
    float lower_bound = bounds[g] - drifts[g];
    bounds[g] = lower_bound;
    if (lower_bound < min_lower_bound) {
      min_lower_bound = lower_bound;
    }
  }
  bounds--;
  // group filter try #1
  if (min_lower_bound >= upper_bound) {
    bounds[0] = upper_bound;
    return;
  }
  upper_bound = 0;
  samples += static_cast<uint64_t>(sample) * features_size;
  uint32_t coffset = cluster * features_size;
  #pragma unroll 4
  for (uint32_t f = 0; f < features_size; f++) {
    float d = samples[f] - centroids[coffset + f];
    upper_bound += d * d;
  }
  upper_bound = sqrt(upper_bound);
  bounds[0] = upper_bound;
  // group filter try #2
  if (min_lower_bound >= upper_bound) {
    return;
  }
  // D'oh!
  passed[atomicAdd(&passed_number, 1)] = sample;
}

__global__ void kmeans_yy_local_filter(
    const uint32_t border, const float *__restrict__ samples,
    const uint32_t *__restrict__ passed, const float *__restrict__ centroids,
    const uint32_t *__restrict__ groups, const float *__restrict__ drifts,
    uint32_t *__restrict__ assignments, float *__restrict__ bounds) {
  uint32_t sample = blockIdx.x * blockDim.x + threadIdx.x;
  if (sample >= border) {
    return;
  }
  sample = passed[sample];
  samples += static_cast<uint64_t>(sample) * features_size;
  bounds += static_cast<uint64_t>(sample) * (yy_groups_size + 1);
  float upper_bound = bounds[0];
  bounds++;
  uint32_t cluster = assignments[sample];
  uint32_t doffset = clusters_size * features_size;
  float min_dist = upper_bound, second_min_dist = FLT_MAX;
  uint32_t nearest = cluster;
  extern __shared__ float shared_centroids[];
  const uint32_t cstep = shmem_size / features_size;
  const uint32_t size_each = cstep / blockDim.x + 1;

  for (uint32_t gc = 0; gc < clusters_size; gc += cstep) {
    uint32_t coffset = gc * features_size;
    __syncthreads();
    if (threadIdx.x * size_each < cstep) {
      for (uint32_t i = 0; i < size_each; i++) {
        uint32_t ci = threadIdx.x * size_each + i;
        uint32_t local_offset = ci * features_size;
        uint32_t global_offset = coffset + local_offset;
        if (global_offset < clusters_size * features_size) {
          #pragma unroll 4
          for (int f = 0; f < features_size; f++) {
            shared_centroids[local_offset + f] = centroids[global_offset + f];
          }
        }
      }
    }
    __syncthreads();

    for (uint32_t c = gc; c < gc + cstep && c < clusters_size; c++) {
      if (c == cluster) {
        continue;
      }
      uint32_t group = groups[c];
      if (group >= yy_groups_size) {
        // this may happen if the centroid is insane (NaN)
        continue;
      }
      float lower_bound = bounds[group];
      if (lower_bound >= upper_bound) {
        if (lower_bound < second_min_dist) {
          second_min_dist = lower_bound;
        }
        continue;
      }
      lower_bound += drifts[group] - drifts[doffset + c];
      if (second_min_dist < lower_bound) {
        continue;
      }
      float dist = 0;
      uint32_t coffset = (c - gc) * features_size;
      #pragma unroll 4
      for (int f = 0; f < features_size; f++) {
        float d = samples[f] - shared_centroids[coffset + f];
        dist += d * d;
      }
      dist = sqrt(dist);
      if (dist < min_dist) {
        second_min_dist = min_dist;
        min_dist = dist;
        nearest = c;
      } else if (dist < second_min_dist) {
        second_min_dist = dist;
      }
    }
  }
  uint32_t nearest_group = groups[nearest];
  uint32_t previous_group = groups[cluster];
  bounds[nearest_group] = second_min_dist;
  if (nearest_group != previous_group) {
    float pb = bounds[previous_group];
    if (pb > upper_bound) {
      bounds[previous_group] = upper_bound;
    }
  }
  bounds[-1] = min_dist;
  if (cluster != nearest) {
    assignments[sample] = nearest;
    atomicAdd(&changed_number, 1);
  }
}

static int check_changed(int iter, float tolerance, uint32_t samples_size,
                         int32_t verbosity, const std::vector<int> &devs) {
  uint32_t overall_changed = 0;
  FOR_ALL_DEVS(
    uint32_t my_changed = 0;
    CUCH(hipMemcpyFromSymbol(&my_changed, HIP_SYMBOL(changed_number), sizeof(my_changed)),
         kmcudaMemoryCopyError);
    overall_changed += my_changed;
  );
  INFO("iteration %d: %" PRIu32 " reassignments\n", iter, overall_changed);
  if (overall_changed <= tolerance * samples_size) {
    return -1;
  }
  assert(overall_changed <= samples_size);
  uint32_t zero = 0;
  FOR_ALL_DEVS(
    CUCH(hipMemcpyToSymbolAsync(HIP_SYMBOL(changed_number), &zero, sizeof(zero)),
         kmcudaMemoryCopyError);
  );
  return kmcudaSuccess;
}

static KMCUDAResult prepare_mem(
    uint32_t samples_size, uint32_t clusters_size, bool resume, int verbosity,
    const std::vector<int> &devs, udevptrs<uint32_t> *ccounts,
    udevptrs<uint32_t> *assignments,  std::vector<uint32_t> *shmem_sizes) {
  uint32_t zero = 0;
  shmem_sizes->clear();
  FOR_ALL_DEVS(
    uint32_t my_shmem_size;
    CUCH(hipMemcpyFromSymbol(&my_shmem_size, HIP_SYMBOL(shmem_size), sizeof(shmem_size)),
         kmcudaMemoryCopyError);
    shmem_sizes->push_back(my_shmem_size * sizeof(uint32_t));
    CUCH(hipMemcpyToSymbolAsync(HIP_SYMBOL(changed_number), &zero, sizeof(zero)),
         kmcudaMemoryCopyError);
    if (!resume) {
      CUCH(hipMemsetAsync(ccounts, 0, clusters_size * sizeof(uint32_t)),
           kmcudaRuntimeError);
      CUCH(hipMemsetAsync(assignments, 0xff, samples_size * sizeof(uint32_t)),
           kmcudaRuntimeError);
    }
  );
  return kmcudaSuccess;
}


extern "C" {

KMCUDAResult kmeans_cuda_setup(uint32_t samples_size_, uint16_t features_size_,
                               uint32_t clusters_size_, uint32_t yy_groups_size_,
                               const std::vector<int> &devs, int32_t verbosity) {
  FOR_ALL_DEVS(
    CUCH(hipMemcpyToSymbol(HIP_SYMBOL(samples_size), &samples_size_, sizeof(samples_size)),
         kmcudaMemoryCopyError);
    CUCH(hipMemcpyToSymbol(HIP_SYMBOL(features_size), &features_size_, sizeof(features_size)),
         kmcudaMemoryCopyError);
    CUCH(hipMemcpyToSymbol(HIP_SYMBOL(clusters_size), &clusters_size_, sizeof(clusters_size)),
         kmcudaMemoryCopyError);
    CUCH(hipMemcpyToSymbol(HIP_SYMBOL(yy_groups_size), &yy_groups_size_, sizeof(yy_groups_size)),
         kmcudaMemoryCopyError);
    hipDeviceProp_t props;
    CUCH(hipGetDeviceProperties(&props, dev), kmcudaRuntimeError);
    int my_shmem_size = static_cast<int>(props.sharedMemPerBlock);
    DEBUG("GPU #%" PRIu32 " has %d bytes of shared memory per block\n",
          dev, my_shmem_size);
    my_shmem_size /= sizeof(uint32_t);
    CUCH(hipMemcpyToSymbol(HIP_SYMBOL(shmem_size), &my_shmem_size, sizeof(my_shmem_size)),
         kmcudaMemoryCopyError);
  );
  return kmcudaSuccess;
}

KMCUDAResult kmeans_cuda_plus_plus(
    uint32_t samples_size, uint32_t features_size, uint32_t cc, int verbosity,
    const std::vector<int> &devs, const udevptrs<float> &samples,
    udevptrs<float> *centroids, udevptrs<float> *dists,
    udevptrs<float> *dev_sums, float *host_dists, float *dist_sum) {
  auto plan = distribute(samples_size, features_size * sizeof(float), devs);
  uint32_t max_len = 0;
  for (auto &p : plan) {
    auto len = std::get<1>(p);
    if (max_len < len) {
      max_len = len;
    }
  }
  CUMEMSET(*dev_sums, 0, max_len / BS_KMPP + 1);
  size_t host_dist_sums_size = samples_size / BS_KMPP + devs.size();
  std::unique_ptr<float[]> host_dist_sums(new float[host_dist_sums_size]);
  memset(host_dist_sums.get(), 0, host_dist_sums_size * sizeof(float));
  uint32_t dist_offset = 0;
  FOR_ALL_DEVSI(
    auto &p = plan[devi];
    auto offset = std::get<0>(p);
    auto length = std::get<1>(p);
    dim3 block(BS_KMPP, 1, 1);
    dim3 grid(std::get<1>(p) / block.x + 1, 1, 1);
    kmeans_plus_plus<<<grid, block, block.x * sizeof(float)>>>(
        length, cc, samples[devi].get() + offset * features_size,
        (*centroids)[devi].get(), (*dists)[devi].get(), (*dev_sums)[devi].get());
    CUCH(hipMemcpyAsync(
        host_dist_sums.get() + dist_offset, (*dev_sums)[devi].get(),
        grid.x * sizeof(float), hipMemcpyDeviceToHost), kmcudaMemoryCopyError);
    CUCH(hipMemcpyAsync(
        host_dists + dist_offset, (*dists)[devi].get(),
        length * sizeof(float), hipMemcpyDeviceToHost), kmcudaMemoryCopyError);
    dist_offset += grid.x;
  );
  SYNC_ALL_DEVS;
  float ds = 0;
  #pragma omp simd reduction(+:ds)
  for (uint32_t i = 0; i < dist_offset; i++) {
    ds += host_dist_sums[i];
  }
  *dist_sum = ds;
  return kmcudaSuccess;
}

KMCUDAResult kmeans_cuda_lloyd(
    float tolerance, uint32_t samples_size, uint32_t clusters_size,
    uint16_t features_size, int32_t verbosity, bool resume,
    const std::vector<int> &devs, const udevptrs<float> &samples,
    udevptrs<float> *centroids, udevptrs<uint32_t> *ccounts,
    udevptrs<uint32_t> *assignments_prev, udevptrs<uint32_t> *assignments,
    int *iterations = nullptr) {
  std::vector<uint32_t> shmem_sizes;
  RETERR(prepare_mem(samples_size, clusters_size, resume, verbosity, devs,
                     ccounts, assignments, &shmem_sizes));
  auto plans = distribute(samples_size, features_size * sizeof(float), devs);
  auto planc = distribute(clusters_size, features_size * sizeof(float), devs);
  dim3 sblock(BS_LL_ASS, 1, 1);
  dim3 cblock(BS_LL_CNT, 1, 1);
  for (int i = 1; ; i++) {
    if (!resume || i > 1) {
      FOR_ALL_DEVSI(
        auto &p = plans[devi];
        auto offset = std::get<0>(p);
        auto length = std::get<1>(p);
        dim3 sgrid(length / sblock.x + 1, 1, 1);
        kmeans_assign_lloyd<<<sgrid, sblock, shmem_sizes[devi]>>>(
            length, samples[devi].get() + offset * features_size,
            (*centroids)[devi].get(), (*assignments_prev)[devi].get() + offset,
            (*assignments)[devi].get() + offset);
        FOR_OTHER_DEVS(
          CUP2P(assignments_prev, offset, length);
          CUP2P(assignments, offset, length);
        );
      );
      int status = check_changed(i, tolerance, samples_size, verbosity, devs);
      if (status < kmcudaSuccess) {
        if (iterations) {
          *iterations = i;
        }
        return kmcudaSuccess;
      }
      if (status != kmcudaSuccess) {
        return static_cast<KMCUDAResult>(status);
      }
    }
    FOR_ALL_DEVSI(
        auto &p = plans[devi];
        auto offset = std::get<0>(p);
        auto length = std::get<1>(p);
        dim3 cgrid(length / cblock.x + 1, 1, 1);
        kmeans_adjust<<<cblock, cgrid, shmem_sizes[devi]>>>(
            length, samples[devi].get(), (*assignments_prev)[devi].get(),
            (*assignments)[devi].get(),
            (*centroids)[devi].get() + offset * features_size,
            (*ccounts)[devi].get() + offset);
        FOR_OTHER_DEVS(
          CUP2P(centroids, offset * features_size, length * features_size);
          CUP2P(ccounts, offset, length);
        );
    );
  }
}

KMCUDAResult kmeans_cuda_yy(
    float tolerance, uint32_t yinyang_groups, uint32_t samples_size_,
    uint32_t clusters_size_, uint16_t features_size, int32_t verbosity,
    const std::vector<int> &devs, const udevptrs<float> &samples,
    udevptrs<float> *centroids, udevptrs<uint32_t> *ccounts,
    udevptrs<uint32_t> *assignments_prev, udevptrs<uint32_t> *assignments,
    udevptrs<uint32_t> *assignments_yy, udevptrs<float> *centroids_yy,
    udevptrs<float> *bounds_yy, udevptrs<float> *drifts_yy,
    udevptrs<uint32_t> *passed_yy) {
  if (yinyang_groups == 0 || YINYANG_DRAFT_REASSIGNMENTS <= tolerance) {
    if (verbosity > 0) {
      if (yinyang_groups == 0) {
        printf("too few clusters for this yinyang_t => Lloyd\n");
      } else {
        printf("tolerance is too high (>= %.2f) => Lloyd\n",
               YINYANG_DRAFT_REASSIGNMENTS);
      }
    }
    return kmeans_cuda_lloyd(
        tolerance, samples_size_, clusters_size_, features_size, verbosity,
        false, devs, samples, centroids, ccounts, assignments_prev, assignments);
  }
  return kmcudaSuccess;
  #if 0
  INFO("running Lloyd until reassignments drop below %" PRIu32 "\n",
       (uint32_t)(YINYANG_DRAFT_REASSIGNMENTS * samples_size_));
  int iter;
  RETERR(kmeans_cuda_lloyd(
      YINYANG_DRAFT_REASSIGNMENTS, samples_size_, clusters_size_, features_size,
      verbosity, false, devs, samples, centroids, ccounts, assignments_prev,
      assignments, &iter));
  if (check_changed(iter, tolerance, samples_size_, 0, devs) < kmcudaSuccess) {
    return kmcudaSuccess;
  }

  // map each centroid to yinyang group -> assignments_yy
  CUCH(hipMemcpyToSymbol(HIP_SYMBOL(samples_size), &clusters_size_, sizeof(samples_size_)),
       kmcudaMemoryCopyError);
  CUCH(hipMemcpyToSymbol(HIP_SYMBOL(clusters_size), &yinyang_groups, sizeof(clusters_size_)),
       kmcudaMemoryCopyError);
  udevptrs<float> tmpbufs, tmpbufs2;
  for (auto &pyy : *passed_yy) {
    tmpbufs.emplace_back(reinterpret_cast<float*>(pyy.get()) +
        samples_size_ - clusters_size_ - yinyang_groups, true);
    tmpbufs2.emplace_back(tmpbufs.back().get() + clusters_size_, true);
  }
  RETERR(kmeans_init_centroids(
      kmcudaInitMethodPlusPlus, clusters_size_, features_size, yinyang_groups,
      0, verbosity, devs, *centroids, &tmpbufs, drifts_yy, centroids_yy),
    INFO("kmeans_init_centroids() failed for yinyang groups: %s\n",
         hipGetErrorString(hipGetLastError())));
  RETERR(kmeans_cuda_lloyd(
      YINYANG_GROUP_TOLERANCE, clusters_size_, yinyang_groups, features_size,
      verbosity, false, devs, *centroids, centroids_yy,
      reinterpret_cast<udevptrs<uint32_t> *>(&tmpbufs2),
      reinterpret_cast<udevptrs<uint32_t> *>(&tmpbufs), assignments_yy));

  CUCH(hipMemcpyToSymbol(HIP_SYMBOL(samples_size), &samples_size_, sizeof(samples_size_)),
       kmcudaMemoryCopyError);
  CUCH(hipMemcpyToSymbol(HIP_SYMBOL(clusters_size), &clusters_size_, sizeof(clusters_size_)),
       kmcudaMemoryCopyError);
  std::vector<uint32_t> shmem_sizes;
  RETERR(prepare_mem(samples_size_, clusters_size_, true, verbosity, devs,
                     ccounts, assignments, &shmem_sizes));
  dim3 siblock(BS_YY_INI, 1, 1);
  dim3 sigrid(samples_size_ / siblock.x + 1, 1, 1);
  dim3 sgblock(BS_YY_GFL, 1, 1);
  dim3 sggrid(samples_size_ / sgblock.x + 1, 1, 1);
  dim3 slblock(BS_YY_LFL, 1, 1);
  dim3 slgrid(samples_size_ / slblock.x + 1, 1, 1);
  dim3 cblock(BS_LL_CNT, 1, 1);
  dim3 cgrid(clusters_size_ / cblock.x + 1, 1, 1);
  dim3 gblock(BLOCK_SIZE, 1, 1);
  dim3 ggrid(yinyang_groups / gblock.x + 1, 1, 1);
  bool refresh = true;
  uint32_t passed_number_ = 0;
  for (; ; iter++) {
    if (!refresh) {
      int status = check_changed(iter, tolerance, samples_size_, verbosity, devs);
      if (status < kmcudaSuccess) {
        return kmcudaSuccess;
      }
      if (status != kmcudaSuccess) {
        return static_cast<KMCUDAResult>(status);
      }
      CUCH(hipMemcpyFromSymbol(&passed_number_, HIP_SYMBOL(passed_number), sizeof(passed_number_)),
           kmcudaMemoryCopyError);
      DEBUG("passed number: %" PRIu32 "\n", passed_number_);
      if (1.f - (passed_number_ + 0.f) / samples_size_ < YINYANG_REFRESH_EPSILON) {
        refresh = true;
      }
      passed_number_ = 0;
    }
    if (refresh) {
      INFO("refreshing Yinyang bounds...\n");
      kmeans_yy_init<<<sigrid, siblock, my_shmem_size>>>(
          samples, centroids, assignments, assignments_yy, bounds_yy);
      refresh = false;
    }
    CUCH(hipMemcpyAsync(
        drifts_yy, centroids, clusters_size_ * features_size * sizeof(float),
        hipMemcpyDeviceToDevice), kmcudaMemoryCopyError);
    kmeans_adjust<<<cblock, cgrid, my_shmem_size>>>(
          samples, assignments_prev, assignments, centroids, ccounts);
    kmeans_yy_calc_drifts<<<cblock, cgrid>>>(centroids, drifts_yy);
    kmeans_yy_find_group_max_drifts<<<gblock, ggrid, my_shmem_size>>>(
        assignments_yy, drifts_yy);
    CUCH(hipMemcpyToSymbolAsync(HIP_SYMBOL(passed_number), &passed_number_, sizeof(passed_number_)),
         kmcudaMemoryCopyError);
    kmeans_yy_global_filter<<<sggrid, sgblock>>>(
        samples, centroids, assignments_yy, drifts_yy, assignments,
        assignments_prev, bounds_yy, passed_yy);
    kmeans_yy_local_filter<<<slgrid, slblock, my_shmem_size>>>(
        samples, passed_yy, centroids, assignments_yy, drifts_yy, assignments,
        bounds_yy);
  }
  #endif
}
}
